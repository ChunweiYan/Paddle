#include <hip/hip_runtime_api.h>
#include <gtest/gtest.h>
#include "basic.h"
#include "math.h"
#include "time.h"

TEST(basic, test0) {
  hipProfilerStop();
  const int dim = 100000;
  float* aD = CreateVec<float>(dim);
  float* bD = CreateVec<float>(dim);
  float* cD = CreateVec<float>(dim);

  float* a = new float[dim];
  float* b = new float[dim];
  float* c = new float[dim];

  RandVec<float>(a, dim);
  RandVec<float>(b, dim);
  RandVec<float>(c, dim);

  const int nbyte = dim * sizeof(float);
  hipMemcpy(aD, a, nbyte, hipMemcpyHostToDevice);
  hipMemcpy(bD, b, nbyte, hipMemcpyHostToDevice);
  hipMemcpy(cD, c, nbyte, hipMemcpyHostToDevice);

  dim3 threadnum{256};
  dim3 block_num{(dim + 1) / threadnum.x};

  hipProfilerStart();
  VecAdd<<<block_num, threadnum>>>(aD, bD, cD, dim);

  hipMemcpy(c, cD, nbyte, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipProfilerStop();
}

TEST(basic, stream) {
  hipStream_t stream0, stream1, stream2;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  const int dim = 1000;
  HdMem<float> A(dim);
  HdMem<float> B(dim);
  HdMem<float> C(dim);

  A.ToDevice();
  B.ToDevice();
  C.ToDevice();

  dim3 threadnum{256};
  dim3 block_num{(dim + 1) / threadnum.x};

  hipProfilerStart();
  VecDot<<<block_num, threadnum, 0, stream0>>>(A.d, B.d, C.d, dim);
  VecDot<<<block_num, threadnum, 0, stream1>>>(A.d, B.d, C.d, dim);
  VecDot<<<block_num, threadnum, 0, stream2>>>(A.d, B.d, C.d, dim);
  hipProfilerStop();
}

TEST(basic, event) {
  hipStream_t stream0, stream1, stream2;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  hipEvent_t event0, event1;
  hipEventCreate(&event0);
  hipEventCreate(&event1);

  const int dim = 1000;
  HdMem<float> A(dim);
  HdMem<float> B(dim);
  HdMem<float> C(dim);

  A.ToDevice();
  B.ToDevice();
  C.ToDevice();

  dim3 threadnum{256};
  dim3 block_num{(dim + 1) / threadnum.x};

  hipProfilerStart();
  VecDot<<<block_num, threadnum, 0, stream0>>>(A.d, B.d, C.d, dim);
  hipEventRecord(event0, stream0);
  VecDot<<<block_num, threadnum, 0, stream1>>>(A.d, B.d, C.d, dim);
  hipEventRecord(event1, stream1);

  // make this kernel wait for the previous two kernel finish.
  hipStreamWaitEvent(stream2, event0, 0);
  hipStreamWaitEvent(stream2, event1, 0);
  VecDot<<<block_num, threadnum, 0, stream2>>>(A.d, B.d, C.d, dim);

  hipProfilerStop();
}

TEST(basic, pinned_memory) {
  const int dim = 10000;
  float* A;
  float* dA;
  hipStream_t stream0, stream1;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);

  // malloc pined memory
  hipHostMalloc(&A, dim * sizeof(float), 0);
  memset(A, 0, dim * sizeof(float));

  // malloc cuda memory
  dA = CreateVec<float>(dim);

  // prepare for a compute kernel
  HdMem<float> a(dim);
  HdMem<float> b(dim);
  HdMem<float> c(dim);
  a.ToDevice();
  b.ToDevice();
  c.ToDevice();

  hipProfilerStart();

  dim3 threadnum{256};
  dim3 block_num{(dim + 1) / threadnum.x};
  // begin parallel
  VecDot<<<threadnum, block_num, 0, stream1>>>(a.d, b.d, c.d, dim);

  for (int i = 0; i < 10; i++) {
    hipMemcpyAsync(dA, A, sizeof(float) * dim, hipMemcpyHostToDevice,
                    stream0);
  }

  hipDeviceSynchronize();
}

TEST(basic, pagable_memory) {
  const int dim = 10000;
  float* A;
  float* dA;
  hipStream_t stream0, stream1;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);

  // malloc pined memory
  A = new float[dim];
  memset(A, 0, dim * sizeof(float));

  // malloc cuda memory
  dA = CreateVec<float>(dim);

  // prepare for a compute kernel
  HdMem<float> a(dim);
  HdMem<float> b(dim);
  HdMem<float> c(dim);
  a.ToDevice();
  b.ToDevice();
  c.ToDevice();

  hipProfilerStart();

  dim3 threadnum{256};
  dim3 block_num{(dim + 1) / threadnum.x};
  // begin parallel
  VecDot<<<threadnum, block_num, 0, stream1>>>(a.d, b.d, c.d, dim);

  for (int i = 0; i < 10; i++) {
    hipMemcpyAsync(dA, A, sizeof(float) * dim, hipMemcpyHostToDevice,
                    stream0);
  }

  hipDeviceSynchronize();
}

TEST(basic, huge_stream_num) {
  const int kStreamNum = 4000;
  hipStream_t streams[kStreamNum];
  for (int i = 0; i < kStreamNum; i++) {
    hipStreamCreate(&streams[i]);
  }

  const int dim = 1000;
  HdMem<float> A(dim);
  HdMem<float> B(dim);
  HdMem<float> C(dim);

  RandVec<float>(A.h, dim);
  RandVec<float>(B.h, dim);
  RandVec<float>(C.h, dim);

  A.ToDevice();
  B.ToDevice();
  C.ToDevice();

  dim3 threadnum{256};
  dim3 block_num{(dim + 1) / threadnum.x};

  hipProfilerStart();
  for (int i = 0; i < kStreamNum; i++) {
    VecDot<<<block_num, threadnum, 0, streams[i]>>>(A.d, B.d, C.d, dim);
  }

  hipDeviceSynchronize();
}

/*
 * 100 kernel works parallelly
 * another 100 kernel wait for several of them
 */
TEST(basic, huge_stream_num_with_event) {
  const int kStreamNum = 200;
  hipStream_t streams[kStreamNum];
  hipEvent_t events[kStreamNum / 2];  // only need half number of events
  for (int i = 0; i < kStreamNum; i++) {
    hipStreamCreate(&streams[i]);
    if (i < kStreamNum / 2) {
      hipEventCreate(&events[i], 0);
    }
  }

  // assign dependency
  int depend_ratio = 0.1;
  std::vector<std::vector<hipEvent_t>> dependencies;
  for (int i = 0; i < kStreamNum / 2; i++) {
    dependencies.emplace_back();
    for (int j = 0; j < kStreamNum / 2; j++) {
      if (1. * rand() / RAND_MAX < depend_ratio) {
        dependencies.back().push_back(events[j]);
      }
    }
  }

  const int dim = 1000;
  HdMem<float> A(dim);
  HdMem<float> B(dim);
  HdMem<float> C(dim);

  RandVec<float>(A.h, dim);
  RandVec<float>(B.h, dim);
  RandVec<float>(C.h, dim);

  A.ToDevice();
  B.ToDevice();
  C.ToDevice();

  dim3 threadnum{256};
  dim3 block_num{(dim + 1) / threadnum.x};

  hipProfilerStart();

  // launch the previous 100 kernels with unique stream each, and they work
  // concurrently.
  for (int i = 0; i < kStreamNum / 2; i++) {
    VecDot<<<block_num, threadnum, 0, streams[i]>>>(A.d, B.d, C.d, dim);
    hipEventRecord(events[i], streams[i]);
  }

  // launch the last left kernels, with random dependency.
  for (int i = kStreamNum / 2; i < kStreamNum; i++) {
    for (auto event : dependencies[i - kStreamNum / 2]) {
      hipStreamWaitEvent(streams[i], event, 0);
    }
    VecDot<<<block_num, threadnum, 0, streams[i]>>>(A.d, B.d, C.d, dim);
  }

  hipDeviceSynchronize();
}

TEST(math, matrix_add) {
  int nx = 100;
  int ny = 200;

  HdMem<int> mem0(nx * ny);
  HdMem<int> mem1(nx * ny);
  HdMem<int> mem2(nx * ny);

  for (int i = 0; i < nx * ny; i++) {
    mem0.h[i] = i;
    mem1.h[i] = i;
  }
  mem0.ToDevice();
  mem1.ToDevice();

  int dx = 3;
  int dy = 3;

  dim3 block(dx, dy);
  dim3 grid((nx + dx - 1) / dx, (ny + dy - 1) / dy);

  SumMatrix<<<grid, block>>>(mem0.d, mem1.d, mem2.d, nx, ny);

  mem2.ToHost();

  hipDeviceSynchronize();

  std::stringstream ss;
  for (int i = 0; i < 100; i++) {
    ss << mem2.h[i] << " ";
  }
  LOG(INFO) << "out " << ss.str();
}
